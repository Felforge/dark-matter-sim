#include <hip/hip_runtime.h>
#include <cmath>


// Copied from go program
struct Particle {
    double Mass;
    double X, Y, Z;
    double Vx, Vy, Vz;
    double Ax, Ay, Az;
};

// atomicAdd does not support doubles by default so this is needed
__device__ double atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void accumulateItemsKernel(Particle* particles, int numParticles, double* dA, double* dD) {
    // Calculate the global thread index
    // Each thread gets a unique "idx" to work on a different particle
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure the thread index is within bounds
    if (idx >= numParticles) {
        return;
    }
    Particle& p = particles[idx];

    double magnitudeA = sqrt(p.Ax * p.Ax + p.Ay * p.Ay + p.Az * p.Az);
    atomicAddDouble(dA, magnitudeA);

    double accumulatedDistance = 0.0;
    for (int i = 0; i < numParticles; i++) {
        if (i == idx) {
            continue;
        }
        double dx = p.X - particles[i].X;
        double dy = p.Y - particles[i].Y;
        double dz = p.Z - particles[i].Z;
        accumulatedDistance += sqrt(dx * dx + dy * dy + dz * dz);
    }
    atomicAddDouble(dD, accumulatedDistance / (numParticles - 1));
}

extern "C" {
    __declspec(dllexport) double getTimeStep(Particle* particles, int numParticles, double timeStepParameter, double softeningDivisor) {
        // Assign number of threads
        int threads = 256;
        // Compute the number of thread blocks needed
        int blocks = (numParticles + threads - 1) / threads;

        // Create device variables
        double *dA, *dD;

        // Allocate memory
        hipMalloc(&dA, sizeof(double));
        hipMalloc(&dD, sizeof(double));

        // Set values
        hipMemset(dA, 0, sizeof(double));
        hipMemset(dD, 0, sizeof(double));
        
        // Accumulate distances and accelerations in parallel on the GPU
        accumulateItemsKernel<<<blocks, threads>>>(particles, numParticles, dA, dD);

        // Wait for everything to finish
        hipDeviceSynchronize();

        // Move items to host
        double hA, hD;
        hipMemcpy(&hA, &dA, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&hD, &dD, sizeof(double), hipMemcpyDeviceToHost);

        // Free device variables from memory
        hipFree(dA);
        hipFree(dD);

        // Average out
        hA /= numParticles;
        hD /= numParticles;

        // Solve for softening length
        double softeningLength = hD / softeningDivisor;

        // Compute time step
        double timeStep = timeStepParameter * sqrt(softeningLength / hA);

        // Return time step
        return timeStep;
    }
}